
#include <hip/hip_runtime.h>
#include <iostream>
void matmul(int *A, int *B, int *C, int N)
{
    for (int Row = 0; Row < N; Row++)
    {
        for (int Col = 0; Col < N; Col++)
        {
            int Pvalue = 0;
            for (int k = 0; k < N; k++)
            {
                Pvalue += A[Row * N + k] * B[k * N + Col];
            }
            C[Row * N + Col] = Pvalue;
        }
    }
}
int main()
{
    int N;
    std::cout<<"\nEnter number of rows: ";
    std::cin>>N;
    int size = N * N * sizeof(int);
    int *A, *B, *C;
    A = new int[N];
    B = new int[N];
    C = new int[N];
    // Initialize matrices A and B
    std::cout<<"\nEnter elements in matrix 1: ";
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cin>>A[i * N + j];
            B[i * N + j] = j * N + i;
        }
    }
    std::cout<<"\nEnter elements in matrix 2: ";
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cin>>B[i * N + j];
        }
    }
    // Perform matrix multiplication
    matmul(A, B, C, N);
    // Print the result (just printing the first 10x10 elements)
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
    // Free memory
    delete[] A;
    delete[] B;
    delete[] C;
    return 0;
}
/*
without gpu
1. sudo apt install nvidia-cuda-toolkit
2. nvcc B190244310.cu -o B190244310
3. ./B190244310
*/
