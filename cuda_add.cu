
#include <hip/hip_runtime.h>
#include <iostream>
void addVectors(int *A, int *B, int *C, int n)
{
    for (int i = 0; i < n; i++)
    {
        C[i] = A[i] + B[i];
    }
}
int main()
{
    int n = 1000000;
    int *A, *B, *C;
    A = new int[n];
    B = new int[n];
    C = new int[n];
    for (int i = 0; i < n; i++)
    {
        A[i] = i;
        B[i] = i * 2;
    }
    addVectors(A, B, C, n);
    for (int i = 0; i < 10; i++)
    {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;
    delete[] A;
    delete[] B;
    delete[] C;
    return 0;
}

/*
without gpu
1. sudo apt install nvidia-cuda-toolkit
2. nvcc B190244310.cu -o B190244310
3. ./B190244310
*/
